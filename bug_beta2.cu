#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

__global__ void kernel(const double *d_X, const double *d_Y, double *res, const int nn, const int pp)
{
  __shared__ double alpha, beta;
  __shared__ int n, p;
  alpha = 1.0; beta = 0.0;
  n = nn; p = pp;

  __shared__ double *d_coef;
  //d_coef = (double*)malloc(sizeof(double)*p*256);
  __syncthreads();
  if (threadIdx.x == 0)
  {
    d_coef = (double*)malloc(sizeof(double)*p*256);
    //memset(d_coef, 0, sizeof(double)*p*256);
  }
  __syncthreads();
  if(!d_coef)
  {
    printf("error\n");
    return;
  }

  //int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = hipblasCreate(&cublasH);

  // X'Y (X is n*p, Y is n*1)
  //double *d_coef = (double*)malloc(sizeof(double)*p);
  //memset(d_coef, 0, sizeof(double)*p);
  //printf("Thread %d got pointer: %p\n", tid, d_coef);
  //double d_coef[3];
  __syncthreads();
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           n, p,
                           &alpha,
                           d_X, n,
                           d_Y, 1,
                           &beta,
                           d_coef+tid*p, 1);
   __syncthreads(); //应该不用加，不存在对share memory和global memory的写入。
  if (cublas_status == HIPBLAS_STATUS_SUCCESS)
   {
     ;
  //   if(d_coef[0]==0)
        printf("Thread %d got pointer: %p, %f, %f, %f\n", tid, d_coef, d_coef[0+tid*p], d_coef[1+tid*p], d_coef[2+tid*p]);
    //printf("tid = %d; d_coef = %f, %f, %f\n", tid, d_coef[0], d_coef[1], d_coef[2]);
    __syncthreads();
   }
   else
   {
     printf("wrong!\n");
     __syncthreads();
   }
   for (size_t i = 0; i < p; i++)
   {
     ;
    // res[tid*p+i] = d_coef[i];
   }
   hipblasDestroy(cublasH);
   //free(d_coef);
   __syncthreads();
   if (threadIdx.x == 0)
    free(d_coef);
}

int main(int argc, char const *argv[]) {
  double A[] = {1, 1, 1, 1, 2, 3, 5, 4, 3, 6, 7, 9};
  double B[] = {1, 2, 3, 4};
  double *d_A, *d_B, *d_res;

  int n = 4, p = 3;
  int threadsPerBlock = 256;
  int blocksPerGird = 40;

  double *res = (double*)malloc(sizeof(double)*p*blocksPerGird*threadsPerBlock);
  //hipDeviceReset();
  size_t limit_stack, limit_printf, limit_heap;
  cudaThreadGetLimit(&limit_stack, hipLimitStackSize);
  cudaThreadGetLimit(&limit_printf, hipLimitPrintfFifoSize);
  cudaThreadGetLimit(&limit_heap, hipLimitMallocHeapSize);
//  cudaThreadSetLimit(hipLimitStackSize, limit*2);
  printf("%d, %d, %d\n", (int)limit_stack, (int)limit_printf, (int)limit_heap);
  cudaThreadSetLimit(hipLimitStackSize, 1024*1024*2);
  cudaThreadSetLimit(hipLimitPrintfFifoSize, 1024*1024*10);
  cudaThreadSetLimit(hipLimitMallocHeapSize, 1024*1024*1024);
  cudaThreadGetLimit(&limit_stack, hipLimitStackSize);
  cudaThreadGetLimit(&limit_printf, hipLimitPrintfFifoSize);
  cudaThreadGetLimit(&limit_heap, hipLimitMallocHeapSize);
//  cudaThreadSetLimit(hipLimitStackSize, limit*2);
  printf("%d, %d, %d\n", (int)limit_stack, (int)limit_printf, (int)limit_heap);
//  cudaThreadGetLimit(&limit, hipLimitMallocHeapSize);
//  printf("%d\n", limit);
  hipMalloc((void**)&d_A, sizeof(double)*n*p);
  hipMalloc((void**)&d_B, sizeof(double)*n);
  hipMalloc((void**)&d_res, sizeof(double)*p*threadsPerBlock*blocksPerGird);
  hipMemcpy(d_A, A, sizeof(double)*n*p, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(double)*n, hipMemcpyHostToDevice);
  kernel<<<blocksPerGird, threadsPerBlock>>>(d_A, d_B, d_res, n, p);
  hipDeviceSynchronize();
  hipMemcpy(res, d_res, sizeof(double)*p*threadsPerBlock*blocksPerGird, hipMemcpyDeviceToHost);
  for (size_t i = 0; i < threadsPerBlock*blocksPerGird; i++)
  {
    ;
    //printf("%f, %f, %f\n", res[i*p], res[i*p+1], res[i*p+2]);
  }
  hipDeviceReset();
  return 0;
}
