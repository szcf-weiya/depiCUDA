#include "hip/hip_runtime.h"
/*
 * Routines for calling cuLUsolve in device
 */

/* Includes, system */
#include <stdio.h>
#include <stdlib.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>


#define PERR(call) \
  if (call) {\
   fprintf(stderr, "%s:%d Error [%s] on "#call"\n", __FILE__, __LINE__,\
      hipGetErrorString(hipGetLastError()));\
   exit(1);\
  }

#define ERRCHECK \
  if (hipPeekAtLastError()) { \
    fprintf(stderr, "%s:%d Error [%s]\n", __FILE__, __LINE__,\
       hipGetErrorString(hipGetLastError()));\
    exit(1);\
  }


__global__ void ols_kernel(const double *d_G,
                            const int n,
                            const int p,
                            double *d_Y,
                            double *d_Gcoef,
                            double *d_Gtscore,
                            const int N)
{
  int id_i = blockIdx.x;
  int id_j = threadIdx.x;
  if (id_j <= id_x)
    return;

  double *d_X1, *d_X2;
  //double *d_X1 = (double*)malloc(sizeof(double)*n);
  //double *d_X2 = (double*)malloc(sizeof(double)*n);
  double *d_X3 = (double*)malloc(sizeof(double)*n);
  d_X1 = d_G + id_i*n;
  d_X2 = d_G + id_j*n;

  // create cublas handle
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = hipblasCreate(&cublasH);
  if (cublas_status != HIPBLAS_STATUS_SUCCESS)
  {
    return;
  }

  // elements-by-elements
  // x3 = x1.*x2
  cublas_status = hipblasDdgmm(cublasH, HIPBLAS_SIDE_LEFT,
                          n, 1,
                          d_X1, n,
                          d_X2, 1,
                          d_X3, n);
  __syncthreads();

  double one = 1.0;
  double *pone = &one;

  // construct matrix X
  double *d_X = (double*)malloc(sizeof(double) * n * p);
  hipblasDcopy(cublasH, n,
                           pone, 0,
                           d_X, 1);
  __syncthreads();
  hipblasDcopy(cublasH, n,
                           d_X1, 1,
                           d_X+n, 1);
  __syncthreads();
  hipblasDcopy(cublasH, n,
                           d_X2, 1,
                           d_X+2*n, 1);
  __syncthreads();
  hipblasDcopy(cublasH, n,
                           d_X3, 1,
                           d_X+3*n, 1);
  __syncthreads();
  free(d_X3);

  // //////////////////
  //
  // X'X
  //
  // /////////////////
  double alpha_v = 1.0;
  double beta_v = 0.0;
  double *alpha = &alpha_v, *beta = &beta_v;
  double *d_XX = (double *)malloc(sizeof(double)*p*p);
  double *d_invXX = (double *)malloc(sizeof(double)*p*p);
  double *d_coef2 = (double *)malloc(sizeof(double)*p);
  double *d_coef = (double *)malloc(sizeof(double)*p);
  double *d_tscore = (double *)malloc(sizeof(double)*p);
  //double *d_Yhat = (double *)malloc(sizeof(double)*n);

  cublas_status = hipblasDgemm(cublasH,
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           p, p, n, // DO NOT mess up the order
                           alpha,
                           d_X, n,
                           d_X, n,
                           beta,
                           d_XX, p);
  __syncthreads();

  // /////////////////////
  //
  // inv(X'X)
  //
  // ////////////////////
  int *pivotArray = (int *)malloc(p*sizeof(int));
  int *info = (int *)malloc(sizeof(int));
  int batch;

  // just one matrix
  info[0] = 0;
  batch = 1;

  double **a = (double **)malloc(sizeof(double *));
  *a = d_XX;
  const double **aconst = (const double **)a;
  double **c = (double **)malloc(sizeof(double *));
  *c = d_invXX;
  cublas_status = hipblasDgetrfBatched(cublasH, p, a, p, pivotArray, info, batch);
  __syncthreads();
  printf("rf %d info %d\n", cublas_status, info[0]);
  cublas_status = hipblasDgetriBatched(cublasH, p, aconst, p, pivotArray,
      c, p, info, batch);
  __syncthreads();
  printf("ri %d info %d\n", cublas_status, info[0]);

  // /////////////////////
  //
  // X'Y   (p*n)*(n*1) = p*1
  //
  // //////////////////////
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           n, p,
                           alpha,
                           d_X, n,
                           d_Y, 1,
                           beta,
                           d_coef2, 1);
  __syncthreads();

  // /////////////////////
  //
  // (X'X)^{-1}X'Y
  //
  // //////////////////////
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           p, p,
                           alpha,
                           d_invXX, p,
                           d_coef2, 1,
                           beta,
                           d_coef, 1);

  __syncthreads();

  // ///////////////////
  // rss
  // ///////////////////

  beta_v = -1.0;
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_N,
                           n, p,
                           alpha,
                           d_X, n,
                           d_coef, 1,
                           beta,
                           d_Y, 1);
  __syncthreads();

  // sigma ^2 = RSS/(n-p-1)
  double sigma;
  printf("%f\n", norm(n, d_Y));
  //sigma = norm(n, d_Y);
  double *psigma = &sigma;
  hipblasDnrm2(cublasH, n, d_Y, 1, psigma);
  //printf("%f\n", sigma);
  sigma = sigma/sqrt((n-p)*1.0);
  for (int i = 0; i < p; i++)
  {
    d_tscore[i] = d_coef[i]/(sigma*sqrt(d_invXX[i+p*i]));
    d_Gcoef[i+id_i*N+id_j-(id_i+1)*(id_i+2)/2] = d_coef[i];
    d_Gtscore[i+id_i*N+id_j-(id_i+1)*(id_i+2)/2] = d_tscore[i];
  }

  printf("%f\n", d_invXX[0]);
  printf("%f\n", d_invXX[1]);
  printf("%f\n", d_invXX[2]);
  printf("%f\n", d_invXX[3]);

  free(d_coef2);
  free(d_coef);
  free(d_tscore);
  free(d_invXX);
  free(d_XX);
  free(d_X);
  //free(d_Yhat);
  free(a);
  free(c);
  hipblasDestroy(cublasH);
}

static void
run_ols(const double *G, const double *Y, int n, int p, double *coef, double *tscore)
{
  double *d_G, *d_Y, *d_coef, *d_tscore;

  int N = 3;

  PERR(hipMalloc(&d_G, n*N*sizeof(double)));
  PERR(hipMalloc(&d_Y, n*sizeof(double)));
  PERR(hipMalloc(&d_coef, N*(N-1)/2*p*sizeof(double)));
  PERR(hipMalloc(&d_tscore, N*(N-1)/2*p*sizeof(double)));
  PERR(hipMemcpy(d_G, G, n*N*sizeof(double), hipMemcpyHostToDevice));
  PERR(hipMemcpy(d_Y, Y, n*sizeof(double), hipMemcpyHostToDevice));


  dim3 blocks(N, 1);
  dim3 grids(N, 1);

  //ols_kernel<<<1, 1>>>(d_X, n, p, d_Y, d_coef, d_tscore);
  ols_kernel<<<grids, blocks>>>(d_X, n, p, d_Y, d_coef, d_tscore, N);

  hipDeviceSynchronize();
  ERRCHECK;

  PERR(hipMemcpy(coef, d_coef, N*(N-1)/2*p*sizeof(double), hipMemcpyDeviceToHost));
  PERR(hipMemcpy(tscore, d_tscore, N*(N-1)/2*p*sizeof(double), hipMemcpyDeviceToHost));

  PERR(hipFree(d_G));
  PERR(hipFree(d_Y));
}

int
main(int argc, char **argv)
{
  double A[] = {1, 3, 4, 5, 2, 3, 5, 4, 3, 6, 7, 9};
  double B[] = {1, 2, 3, 4};
  double coef[2];
  double pvalue[2];

  run_ols(A, B, 4, 4, coef, pvalue);

  printf("beta0 = %f; pvalue = %f\n", coef[0], pvalue[0]);
  printf("beta1 = %f; pvalue = %f\n", coef[1], pvalue[1]);
  return 0;
}
