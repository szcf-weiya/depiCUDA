#include "hip/hip_runtime.h"
/*
 * Routines for calling cuLUsolve in device
 */

/* Includes, system */
#include <stdio.h>
#include <stdlib.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>


#define PERR(call) \
  if (call) {\
   fprintf(stderr, "%s:%d Error [%s] on "#call"\n", __FILE__, __LINE__,\
      hipGetErrorString(hipGetLastError()));\
   exit(1);\
  }

#define ERRCHECK \
  if (hipPeekAtLastError()) { \
    fprintf(stderr, "%s:%d Error [%s]\n", __FILE__, __LINE__,\
       hipGetErrorString(hipGetLastError()));\
    exit(1);\
  }


__global__ void ols_kernel(const double *d_X,
                            const int n,
                            const int p,
                            double *d_Y,
                            double *d_coef,
                            double *d_tscore)
{
  // create cublas handle
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = hipblasCreate(&cublasH);
  if (cublas_status != HIPBLAS_STATUS_SUCCESS)
  {
    return;
  }
  // //////////////////
  //
  // X'X
  //
  // /////////////////
  double alpha_v = 1.0;
  double beta_v = 0.0;
  double *alpha = &alpha_v, *beta = &beta_v;
  double *d_XX = (double *)malloc(sizeof(double)*p*p);
  double *d_invXX = (double *)malloc(sizeof(double)*p*p);
  double *d_coef2 = (double *)malloc(sizeof(double)*p);
  //double *d_Yhat = (double *)malloc(sizeof(double)*n);

  cublas_status = hipblasDgemm(cublasH,
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           p, p, n, // DO NOT mess up the order
                           alpha,
                           d_X, n,
                           d_X, n,
                           beta,
                           d_XX, p);
  __syncthreads();

  // /////////////////////
  //
  // inv(X'X)
  //
  // ////////////////////
  int *pivotArray = (int *)malloc(p*sizeof(int));
  int *info = (int *)malloc(sizeof(int));
  int batch;

  // just one matrix
  info[0] = 0;
  batch = 1;

  double **a = (double **)malloc(sizeof(double *));
  *a = d_XX;
  const double **aconst = (const double **)a;
  double **c = (double **)malloc(sizeof(double *));
  *c = d_invXX;
  cublas_status = hipblasDgetrfBatched(cublasH, p, a, p, pivotArray, info, batch);
  __syncthreads();
  printf("rf %d info %d\n", cublas_status, info[0]);
  cublas_status = hipblasDgetriBatched(cublasH, p, aconst, p, pivotArray,
      c, p, info, batch);
  __syncthreads();
  printf("ri %d info %d\n", cublas_status, info[0]);

  // /////////////////////
  //
  // X'Y   (p*n)*(n*1) = p*1
  //
  // //////////////////////
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           n, p,
                           alpha,
                           d_X, n,
                           d_Y, 1,
                           beta,
                           d_coef2, 1);
  __syncthreads();

  // /////////////////////
  //
  // (X'X)^{-1}X'Y
  //
  // //////////////////////
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           p, p,
                           alpha,
                           d_invXX, p,
                           d_coef2, 1,
                           beta,
                           d_coef, 1);

  __syncthreads();

  // ///////////////////
  // rss
  // ///////////////////

  beta_v = -1.0;
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_N,
                           n, p,
                           alpha,
                           d_X, n,
                           d_coef, 1,
                           beta,
                           d_Y, 1);
  __syncthreads();

  // sigma ^2 = RSS/(n-p-1)
  double sigma;
  double *psigma = &sigma;
  hipblasDnrm2(cublasH, n, d_Y, 1, psigma);
  //sigma = norm(n, d_Y);
  printf("%f\n", sigma);
  sigma = sigma/sqrt((n-p)*1.0);
  for (int i = 0; i < p; i++)
    d_tscore[i] = d_coef[i]/(sigma*sqrt(d_invXX[i+p*i]));
  printf("%f\n", d_invXX[0]);
  printf("%f\n", d_invXX[1]);
  printf("%f\n", d_invXX[2]);
  printf("%f\n", d_invXX[3]);

  free(d_coef2);
  free(d_invXX);
  free(d_XX);
  //free(d_Yhat);
  free(a);
  free(c);
  hipblasDestroy(cublasH);
}

static void
run_ols(const double *X, const double *Y, int n, int p, double *coef, double *tscore)
{
  double *d_X, *d_Y, *d_coef, *d_tscore;

  PERR(hipMalloc(&d_X, n*p*sizeof(double)));
  PERR(hipMalloc(&d_Y, n*sizeof(double)));
  PERR(hipMalloc(&d_coef, p*sizeof(double)));
  PERR(hipMalloc(&d_tscore, p*sizeof(double)));
  PERR(hipMemcpy(d_X, X, n*p*sizeof(double), hipMemcpyHostToDevice));
  PERR(hipMemcpy(d_Y, Y, n*sizeof(double), hipMemcpyHostToDevice));

  ols_kernel<<<1, 1>>>(d_X, n, p, d_Y, d_coef, d_tscore);

  hipDeviceSynchronize();
  ERRCHECK;

  PERR(hipMemcpy(coef, d_coef, p*sizeof(double), hipMemcpyDeviceToHost));
  PERR(hipMemcpy(tscore, d_tscore, p*sizeof(double), hipMemcpyDeviceToHost));

  PERR(hipFree(d_X));
  PERR(hipFree(d_Y));
}

int
main(int argc, char **argv)
{
  double A[] = {1, 1, 1, 1, 2, 3, 5, 4};
  double B[] = {1, 2, 3, 4};
  double coef[2];
  double pvalue[2];

  run_ols(A, B, 4, 2, coef, pvalue);

  printf("beta0 = %f; pvalue = %f\n", coef[0], pvalue[0]);
  printf("beta1 = %f; pvalue = %f\n", coef[1], pvalue[1]);
  return 0;
}
