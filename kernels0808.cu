#include "hip/hip_runtime.h"
/*
 * Routines for calling cuLUsolve in device
 */

/* Includes, system */
#include <stdio.h>
#include <stdlib.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <gsl/gsl_rng.h>
#include <gsl/gsl_matrix.h>


#define PERR(call) \
  if (call) {\
   fprintf(stderr, "%s:%d Error [%s] on "#call"\n", __FILE__, __LINE__,\
      hipGetErrorString(hipGetLastError()));\
   exit(1);\
  }

#define ERRCHECK \
  if (hipPeekAtLastError()) { \
    fprintf(stderr, "%s:%d Error [%s]\n", __FILE__, __LINE__,\
       hipGetErrorString(hipGetLastError()));\
    exit(1);\
  }


__global__ void ols_kernel(const double *d_GX,
                            const int n,
                            const int p,
                            const double *d_GY,
                            double *d_Gcoef,
                            double *d_Gtscore,
                            const int N)
{
  int id_i = blockIdx.x;
  int noOfBlocks = (N + blockDim.x - 1)/blockDim.x;
  int id_j;

  // create cublas handle
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = hipblasCreate(&cublasH);

  const double *d_X1, *d_X2;
  double *d_X3 = (double*)malloc(sizeof(double)*n);

  int *pivotArray = (int *)malloc(p*sizeof(int));
  int *info = (int *)malloc(sizeof(int));
  int batch;
  double *pone = (double*)malloc(sizeof(double));
  *pone = 1.0;
  // just one matrix
  info[0] = 0;
  batch = 1;

  double *d_X = (double*)malloc(sizeof(double) * n * p);
  double *d_Y = (double*)malloc(sizeof(double)*n);

  double *d_XX = (double *)malloc(sizeof(double)*p*p);
  double *d_invXX = (double *)malloc(sizeof(double)*p*p);
  double *d_coef2 = (double *)malloc(sizeof(double)*p);
  double *d_coef = (double *)malloc(sizeof(double)*p);

  double sigma;
  double *psigma = (double*)malloc(sizeof(double));
  int id;
  double tscore;

  double alpha_v = 1.0;
  double beta_v = 0.0;
  double *alpha = &alpha_v, *beta = &beta_v;

  double **a = (double **)malloc(sizeof(double *));
  *a = d_XX;
  const double **aconst = (const double **)a;

  double **c = (double **)malloc(sizeof(double *));
  *c = d_invXX;


  __syncthreads();
  for (int id_ii = 0; id_ii < noOfBlocks; id_ii++)
  {
    id_j = threadIdx.x + id_ii*blockDim.x;
    if (id_j > id_i && id_j < N)
    {
    d_X1 = d_GX + id_i*n;
    d_X2 = d_GX + id_j*n;
    __syncthreads();

    // elements-by-elements
    // x3 = x1.*x2
    cublas_status = hipblasDdgmm(cublasH, HIPBLAS_SIDE_LEFT,
                            n, 1,
                            d_X1, n,
                            d_X2, 1,
                            d_X3, n);
    __syncthreads();

    // copy to d_Y
    cublas_status = hipblasDcopy(cublasH, n,
                             d_GY, 1,
                             d_Y, 1);
    __syncthreads();


    // construct matrix X
    cublas_status = hipblasDcopy(cublasH, n,
                             pone, 0,
                             d_X, 1);
    __syncthreads();

    cublas_status = hipblasDcopy(cublasH, n,
                             d_X1, 1,
                             d_X+n, 1);
    __syncthreads();

    cublas_status = hipblasDcopy(cublasH, n,
                             d_X2, 1,
                             d_X+2*n, 1);
    __syncthreads();

    cublas_status = hipblasDcopy(cublasH, n,
                             d_X3, 1,
                             d_X+3*n, 1);
    __syncthreads();


    // //////////////////
    //
    // X'X
    //
    // /////////////////
    cublas_status = hipblasDgemm(cublasH,
                             HIPBLAS_OP_T, HIPBLAS_OP_N,
                             p, p, n, // DO NOT mess up the order
                             alpha,
                             d_X, n,
                             d_X, n,
                             beta,
                             d_XX, p);
    __syncthreads();
    // /////////////////////
    //
    // inv(X'X)
    //
    // ////////////////////

    cublas_status = hipblasDgetrfBatched(cublasH, p, a, p, pivotArray, info, batch);
    __syncthreads();
    if (info[0] < 0)
    {
      printf("i = %d, j = %d, in LU decomposition, the %d parameter had an illegeal value\n", id_i, id_j, abs(info[0]));
      continue;
      //return;
    }
    else if (info[0] > 0)
    {
      printf("i = %d, j = %d, in LU decomposition, U(%d, %d) = 0\n", id_i, id_j, abs(info[0]), abs(info[0]));
      continue;
      //return;
    }
    cublas_status = hipblasDgetriBatched(cublasH, p, aconst, p, pivotArray,
        c, p, info, batch);
    if (info[0] < 0)
    {
      printf("i = %d, j = %d, in LU decomposition, the %d parameter had an illegeal value\n", id_i, id_j, abs(info[0]));
      continue;
      //return;
    }
    __syncthreads();


    // /////////////////////
    //
    // X'Y   (p*n)*(n*1) = p*1
    //
    // //////////////////////
    cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                             n, p,
                             alpha,
                             d_X, n,
                             d_Y, 1,
                             beta,
                             d_coef2, 1);
    __syncthreads();
    //printf("%d, %d, finish X'Y\n", id_i, id_j);
    // /////////////////////
    //
    // (X'X)^{-1}X'Y
    //
    // //////////////////////
    cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                             p, p,
                             alpha,
                             d_invXX, p,
                             d_coef2, 1,
                             beta,
                             d_coef, 1);
    __syncthreads();
    //printf("%d, %d, finish beta\n", id_i, id_j);
    // ///////////////////
    // rss
    // ///////////////////
    *beta = -1.0;
    cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_N,
                             n, p,
                             alpha,
                             d_X, n,
                             d_coef, 1,
                             beta,
                             d_Y, 1);
    __syncthreads();
    //printf("%d, %d, finish rss\n", id_i, id_j);
    // sigma ^2 = RSS/(n-p-1)
    //psigma = &sigma;
    cublas_status = hipblasDnrm2(cublasH, n, d_Y, 1, psigma);
    __syncthreads();
    sigma = *psigma;

    //printf("%f\n", sigma);
    sigma = sigma/sqrt((n-p)*1.0);
    __syncthreads();
    for (int i = 0; i < p; i++)
    {
      tscore = d_coef[i]/(sigma*sqrt(d_invXX[i+p*i]));
      id = id_i*N+id_j-((id_i+1)*(id_i+2))/2;
      d_Gcoef[i+p*id] = d_coef[i];
      d_Gtscore[i+p*id] = tscore;
    }

    //printf("%d, %d, finish tscore\n", id_i, id_j);
    printf("i = %d, j = %d; beta = %f, %f, %f, %f\n", id_i, id_j, d_coef[0], d_coef[1], d_coef[2], d_coef[3]);
    //printf("i = %d, j = %d; tscore = %f, %f, %f, %f\n", id_i, id_j, d_tscore[0], d_tscore[1], d_tscore[2], d_tscore[3]);
    }
  }
  free(pone);
  free(d_coef2);
  free(d_coef);
  free(d_invXX);
  free(d_X);
  free(d_Y);
  free(pivotArray); // DO NOT free before
  free(info); // DO NOT free before
  free(d_XX);
  free(a);
  free(psigma);
  free(c); // DO NOT free before d_invXX
  free(d_X3);
  hipblasDestroy(cublasH);
}

static void
run_ols(const double *G, const double *Y, int n, int p, double *coef, double *tscore, int N)
{
  double *d_G, *d_Y, *d_coef, *d_tscore;

  PERR(hipMalloc(&d_G, n*N*sizeof(double)));
  PERR(hipMalloc(&d_Y, n*sizeof(double)));
  PERR(hipMalloc(&d_coef, (N*(N-1))/2*p*sizeof(double)));
  PERR(hipMalloc(&d_tscore, (N*(N-1))/2*p*sizeof(double)));
  PERR(hipMemcpy(d_G, G, n*N*sizeof(double), hipMemcpyHostToDevice));
  PERR(hipMemcpy(d_Y, Y, n*sizeof(double), hipMemcpyHostToDevice));

  int threadsPerBlock = 4;
  int blocksPerGird = N;
//  dim3 blocks(threadsPerBlock, 1);
//  dim3 grids(blocksPerGird, 1);
  //int blocks = N;
  //int grids = N;
  //dim3 blocks(16, 16);
  //dim3 grids((N+15)/16,(N+15)/16);
  //int numBlocks = (N+15)/16;
  //ols_kernel<<<1, 1>>>(d_X, n, p, d_Y, d_coef, d_tscore);
  ols_kernel<<<blocksPerGird, threadsPerBlock>>>(d_G, n, p, d_Y, d_coef, d_tscore, N);
  hipDeviceSynchronize();
  ERRCHECK;

  PERR(hipMemcpy(coef, d_coef, (N*(N-1))/2*p*sizeof(double), hipMemcpyDeviceToHost));
  PERR(hipMemcpy(tscore, d_tscore, (N*(N-1))/2*p*sizeof(double), hipMemcpyDeviceToHost));

  PERR(hipFree(d_G));
  PERR(hipFree(d_Y));
  PERR(hipFree(d_coef));
  PERR(hipFree(d_tscore));
}

int
main(int argc, char **argv)
{
  const gsl_rng_type *T;
  gsl_rng *r;
  gsl_rng_env_setup();
  T = gsl_rng_default;
  r = gsl_rng_alloc(T);

  int N, n, p = 4;
  N = 3;
  n = 4;
  /*
  N = 4;
  n = 305;

  double *A = (double*)malloc(sizeof(double)*n*N);
  double *B = (double*)malloc(sizeof(double)*n);
  double *pvalue = (double*)malloc(sizeof(double)*(N*(N-1))/2*p);
  double *coef = (double*)malloc(sizeof(double)*(N*(N-1))/2*p);
  if (!A)
    printf("pvalue malloc error");
  if (!B)
    printf("pvalue malloc error");

  if (!pvalue)
    printf("pvalue malloc error");
  if (!coef)
    printf("coef malloc error");
  */
  /*
  for (size_t i = 0; i < n; i++)
  {
    for (size_t j = 0; j < N; j++)
      A[j + i*N] = gsl_rng_uniform(r);
    B[i] = gsl_rng_uniform(r);
  }
  */


  double A[] = {1, 3, 4, 5, 2, 3, 5, 4, 3, 6, 7, 9};
  double B[] = {1, 2, 3, 4};
  double coef[4*3];
  double pvalue[4*3];

  run_ols(A, B, n, p, coef, pvalue, N);
  /*
  gsl_matrix_view m = gsl_matrix_view_array(coef,(N*(N-1))/2, p);
  free(A);
  free(B);
  free(pvalue);
  free(coef);
  */
  printf("beta0 = %f; pvalue = %f\n", coef[0], pvalue[0]);
  printf("beta1 = %f; pvalue = %f\n", coef[1], pvalue[1]);
  printf("beta2 = %f; pvalue = %f\n", coef[2], pvalue[2]);
  printf("beta3 = %f; pvalue = %f\n", coef[3], pvalue[3]);

  printf("beta0 = %f; pvalue = %f\n", coef[4], pvalue[4]);
  printf("beta1 = %f; pvalue = %f\n", coef[5], pvalue[5]);
  printf("beta2 = %f; pvalue = %f\n", coef[6], pvalue[6]);
  printf("beta3 = %f; pvalue = %f\n", coef[7], pvalue[7]);

  printf("beta0 = %f; pvalue = %f\n", coef[8], pvalue[8]);
  printf("beta1 = %f; pvalue = %f\n", coef[9], pvalue[9]);
  printf("beta2 = %f; pvalue = %f\n", coef[10], pvalue[10]);
  printf("beta3 = %f; pvalue = %f\n", coef[11], pvalue[11]);


  return 0;
}
