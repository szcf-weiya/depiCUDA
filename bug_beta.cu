#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

__device__ inline int test_mv(int tid, const double *d_X, const double *d_Y, double *d_coef, const int n, const int p)
{
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = hipblasCreate(&cublasH);
  double alpha = 1.0, beta = 0.0;
  // X'Y (X is n*p, Y is n*1)

  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           n, p,
                           &alpha,
                           d_X, n,
                           d_Y, 1,
                           &beta,
                           d_coef, 1);
/*
   cublas_status = hipblasDgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                            p, 1, n,
                            &alpha,
                            d_X, n,
                            d_Y, n,
                            &beta,
                            d_coef, p);
*/
   if (cublas_status == HIPBLAS_STATUS_SUCCESS)
    printf("tid = %d; d_coef = %f, %f, %f\n", tid, d_coef[0], d_coef[1], d_coef[2]);
   else
    printf("wrong!\n");
   hipblasDestroy(cublasH);
   return 0;
}

__global__ void kernel(const double *d_X, const double *d_Y, const int n, const int p)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  double *d_coef = (double*)malloc(sizeof(double)*p);
  test_mv(tid, d_X, d_Y, d_coef, n, p);
}


int main(int argc, char const *argv[]) {
  double A[] = {1, 1, 1, 1, 2, 3, 5, 4, 3, 6, 7, 9};
  double B[] = {1, 2, 3, 4};
  double *d_A, *d_B;
  int n = 4, p = 3;
  int threadsPerBlock = 64;
  int blocksPerGird = 2;
  hipMalloc((void**)&d_A, sizeof(double)*n*p);
  hipMalloc((void**)&d_B, sizeof(double)*n);
  hipMemcpy(d_A, A, sizeof(double)*n*p, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(double)*n, hipMemcpyHostToDevice);
  kernel<<<blocksPerGird, threadsPerBlock>>>(d_A, d_B, n, p);
  hipDeviceReset();
  return 0;
}
