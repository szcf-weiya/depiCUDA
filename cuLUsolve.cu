#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hipsolver.h>

#include "cuLUsolve.h"

// solve Ax = B
//
// where dim(A) = m*m
//       dim(B) = m*1
int cuda_LU_solve(const double *A, const int m, double *B, const int nrhs)
{
  hipsolverHandle_t cusolverH = NULL;
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
  hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
  hipError_t cudaStat1 = hipSuccess;
  hipError_t cudaStat2 = hipSuccess;
  hipError_t cudaStat3 = hipSuccess;
  hipError_t cudaStat4 = hipSuccess;

  const int lda = m;
  const int ldb = m;
  //const int nrhs = 1;
  //
  //     | 1 2 3 |
  // A = | 4 5 6 |
  //     | 2 1 1 |
  //
  // x = (1, 1, 1)'
  // b = (6, 15, 4)'
  //

  //double A[lda*m] = {1, 4, 2, 2, 5, 1, 3, 6, 1};
  //double B[ldb*nrhs] = {6, 15, 4};

  double *d_A = NULL; //linear memory of GPU
  int *devIpiv = NULL;
  double *d_B = NULL;
  int *devInfo = NULL;
  double *d_work = NULL;
  int lwork = 0;
  int info_gpu = 0;

  // step 1: create cusolver/cublas handle
  cusolver_status = hipsolverDnCreate(&cusolverH);
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

  cublas_status = hipblasCreate(&cublasH);
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

  // step 2: copy A and B to device
  cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double) * lda * m);
  cudaStat2 = hipMalloc ((void**)&devIpiv, sizeof(int) * m);
  cudaStat3 = hipMalloc ((void**)&d_B  , sizeof(double) * ldb * nrhs);
  cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);
  assert(hipSuccess == cudaStat3);
  assert(hipSuccess == cudaStat4);

  cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m   , hipMemcpyHostToDevice);
  cudaStat2 = hipMemcpy(d_B, B, sizeof(double) * ldb * nrhs, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);

  // step 3: query working space of getrf and getrs
  cusolver_status = hipsolverDnDgetrf_bufferSize(cusolverH,
                      m,
                      m,
                      d_A,
                      lda,
                      &lwork );

  assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
  cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
  assert(hipSuccess == cudaStat1);

  // step 4: compute LU decomposition
  cusolver_status = hipsolverDnDgetrf(cusolverH,
           m,
           m,
           d_A,
           lda,
           d_work,
           devIpiv,
           devInfo );

  cudaStat1 = hipDeviceSynchronize();
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
  assert(hipSuccess == cudaStat1);

  // check if LU is good or not
  cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);
  printf("after getrf: info_gpu = %d\n", info_gpu);
  if (info_gpu < 0)
    printf("ERROR: the %d-th parameter is wrong.\n", abs(info_gpu));
  else if (info_gpu > 0)
    printf("ERROR: U(%d, %d) = 0\n", info_gpu, info_gpu);

  assert(0 == info_gpu);

  // setp 5: solve Ax = B
  cusolver_status = hipsolverDnDgetrs(cusolverH,
           HIPBLAS_OP_N,
           m,
           nrhs,
           d_A,
           lda,
           devIpiv,
           d_B,
           ldb,
           devInfo );

  cudaStat1 = hipDeviceSynchronize();
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
  assert(hipSuccess == cudaStat1);

  // check if LU is good or not
  cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);

  printf("after getrs: info_gpu = %d\n", info_gpu);
  if (info_gpu < 0)
    printf("ERROR: the %d-th parameter is wrong.\n", abs(info_gpu));

  assert(0 == info_gpu);


  cudaStat1 = hipMemcpy(B, d_B, sizeof(double)*ldb*nrhs, hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);

  //printf("X = (matlab base-1)\n");
  //printMatrix(m, nrhs, B, ldb, "X");

  // free resources
  if (d_A) hipFree(d_A);
  if (devIpiv) hipFree(devIpiv);
  if (d_B) hipFree(d_B);
  if (devInfo) hipFree(devInfo);
  if (d_work) hipFree(d_work);

  if (cublasH ) hipblasDestroy(cublasH);
  if (cusolverH) hipsolverDnDestroy(cusolverH);

  //hipDeviceReset();

  return 0;
}
