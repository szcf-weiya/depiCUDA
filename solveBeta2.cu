#include "hip/hip_runtime.h"

/* Includes, system */
#include <stdio.h>
#include <stdlib.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <gsl/gsl_rng.h>
#include <gsl/gsl_matrix.h>

__device__ inline int solveBeta(int tid,
                          int n, int p,
                          const double *d_X,
                          const double *d_Y,
                          double *d_invXX,
                          double *d_coef)
{
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = hipblasCreate(&cublasH);
  int *pivotArray = (int *)malloc(p*sizeof(int));
  int *info = (int *)malloc(sizeof(int));
  info[0] = 0;
  //int info;
  int batch = 1;
  double alpha = 1, beta = 0;
  const double *palpha = &alpha, *pbeta = &beta;

  double *d_XX = (double *)malloc(sizeof(double)*p*p);
  double *d_coef2 = (double *)malloc(sizeof(double)*p);
  memset(d_coef2, 0.0, sizeof(double)*p);
  memset(d_coef, 0.0, sizeof(double)*p);
  memset(d_XX, 0.0, sizeof(double)*p*p);


  double **a = (double **)malloc(sizeof(double *));
  *a = d_XX;
  const double **aconst = (const double **)a;

  double **c = (double **)malloc(sizeof(double *));
  *c = d_invXX;
  // X'X
  cublas_status = hipblasDgemm(cublasH,
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           p, p, n, // DO NOT mess up the order
                           palpha,
                           d_X, n,
                           d_X, n,
                           pbeta,
                           d_XX, p);

  // inv(X'X)
  cublas_status = hipblasDgetrfBatched(cublasH, p, a, p, pivotArray, info, batch);
  if (info[0] < 0)
  {
    hipblasDestroy(cublasH);
    return info[0];
  }
  else if (info[0] > 0)
  {
    hipblasDestroy(cublasH);
    return info[0];
  }
  cublas_status = hipblasDgetriBatched(cublasH, p, aconst, p, pivotArray, c, p, info, batch);
  if (info[0] < 0)
  {
    hipblasDestroy(cublasH);
    return info[0];
  }
  // X'Y   (p*n)*(n*1) = p*1
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           n, p,
                           palpha,
                           d_X, n,
                           d_Y, 1,
                           pbeta,
                           d_coef2, 1);
  __syncthreads();
  printf("tid = %d; inv = %.4f, %.4f, %.4f, %.4f, %.4f, %.4f, %.4f, %.4f, %.4f\n", tid, d_invXX[0], d_invXX[1], d_invXX[2], d_invXX[5], d_invXX[4], d_invXX[8], d_coef2[0], d_coef2[1], d_coef2[2]);

  // (X'X)^{-1}X'Y
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_N,
                           p, p,
                           palpha,
                           d_invXX, p,
                           d_coef2, 1,
                           pbeta,
                           d_coef, 1);
  //printf("tid = %d; inv = %f, %f, %f; beta = %f, %f, %f\n", tid, d_invXX[0], d_invXX[4], d_invXX[8], d_coef[0], d_coef[1], d_coef[2]);
  //printf("tid = %d; inv = %.4f, %.4f, %.4f, %.4f, %.4f, %.4f, %.4f, %.4f, %.4f\n", tid, d_invXX[0], d_invXX[1], d_invXX[2], d_invXX[5], d_invXX[4], d_invXX[8], d_coef[0], d_coef[1], d_coef[2]);
//  printf("tid = %d; inv = %.4f, %.4f, %.4f, %.4f, %.4f, %.4f, %.4f, %.4f, %.4f\n", tid, d_invXX[0], d_invXX[1], d_invXX[2], d_invXX[5], d_invXX[4], d_invXX[8], d_coef2[0], d_coef2[1], d_coef2[2]);

  free(pivotArray);
  free(info);
  free(d_XX);
  free(d_coef2);
  free(a);
  free(c);
  hipblasDestroy(cublasH);
  return 0;
}

__global__ void kernel(int n, int p,
  const double *d_X,
  const double *d_Y,
  double *d_GinvXX,
  double *d_Gcoef)
{
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  //printf("%d\n", tid);
  double *d_coef = (double*)malloc(sizeof(double)*p);
  double *d_invXX = (double*)malloc(sizeof(double)*p*p);
  //__syncthreads(); // must add
  solveBeta(tid, n, p, d_X, d_Y, d_invXX, d_coef);
  //__syncthreads(); // must add
  /*
  if (res != 0)
  {
    free(d_coef);
    free(d_invXX);
    return;
  }
  */
  //__syncthreads(); // must add
  #pragma unroll
  for (int i = 0; i < p; i++)
  {
    d_Gcoef[tid*p+i] = d_coef[i];
  }
  //printf("tid = %d; inv = %f, %f, %f; beta = %f, %f, %f\n", tid, d_invXX[0], d_invXX[4], d_invXX[8], d_coef[0], d_coef[1], d_coef[2]);
  //__syncthreads();
  free(d_coef);
  free(d_invXX);
}

int main(int argc, char const *argv[]) {
  /* code */
  //double A[] = {1, 2, 3, 0, 2, 4, 2, 1, 5};
  double A[] = {1, 1, 1, 1, 2, 3, 5, 4, 3, 6, 7, 9};
  double B[] = {1, 2, 3, 4};
  double *d_A, *d_B, *d_invXX, *d_coef;
  int threadsPerBlock = 32;
  int blocksPerGird = 1;
  // Set a heap size of 128 megabytes. Note that this must
  // be done before any kernel is launched.
  cudaThreadSetLimit(hipLimitMallocHeapSize, 1024*1024*1024);
  double coef[3*threadsPerBlock*blocksPerGird];
  hipMalloc((void**)&d_A, sizeof(double)*12);
  hipMalloc((void**)&d_B, sizeof(double)*4);
  hipMalloc((void**)&d_invXX, sizeof(double)*9);
  hipMalloc((void**)&d_coef, sizeof(double)*3*threadsPerBlock*blocksPerGird);
  hipMemcpy(d_A, A, sizeof(double)*12, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(double)*4, hipMemcpyHostToDevice);
  kernel<<<threadsPerBlock, blocksPerGird>>>(4, 3, d_A, d_B, d_invXX, d_coef);
  hipDeviceSynchronize();
  hipMemcpy(coef, d_coef, sizeof(double)*3*threadsPerBlock*blocksPerGird, hipMemcpyDeviceToHost);

  hipFree(d_coef);
  hipFree(d_invXX);
  hipFree(d_A);
  hipFree(d_B);
  hipDeviceReset();
/*
  for (int i = 0; i < threadsPerBlock*blocksPerGird; i++)
  {
    for (int j = 0; j < 3; j++)
      printf("%f, \n", coef[i*3+j]);
    printf("\n");
  }
  */
  return 0;
}
