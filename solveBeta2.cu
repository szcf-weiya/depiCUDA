#include "hip/hip_runtime.h"

/* Includes, system */
#include <stdio.h>
#include <stdlib.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <gsl/gsl_rng.h>
#include <gsl/gsl_matrix.h>

__device__ int solveBeta(
                          int n, int p,
                          const double *d_X,
                          const double *d_Y,
                          double *d_invXX,
                          double *d_coef)
{
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = hipblasCreate(&cublasH);
  int *pivotArray = (int *)malloc(p*sizeof(int));
  int *info = (int *)malloc(sizeof(int));
  info[0] = 0;
  //int info;
  int batch = 1;
  double alpha = 1.0, beta = 0.0;

  double *d_XX = (double *)malloc(sizeof(double)*p*p);
  double *d_coef2 = (double *)malloc(sizeof(double)*p);

  double **a = (double **)malloc(sizeof(double *));
  *a = d_XX;
  const double **aconst = (const double **)a;

  double **c = (double **)malloc(sizeof(double *));
  *c = d_invXX;
  // X'X
  cublas_status = hipblasDgemm(cublasH,
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           p, p, n, // DO NOT mess up the order
                           &alpha,
                           d_X, n,
                           d_X, n,
                           &beta,
                           d_XX, p);

  // inv(X'X)
  cublas_status = hipblasDgetrfBatched(cublasH, p, a, p, pivotArray, info, batch);
  if (info[0] < 0)
  {
    hipblasDestroy(cublasH);
    return info[0];
  }
  else if (info[0] > 0)
  {
    hipblasDestroy(cublasH);
    return info[0];
  }
  cublas_status = hipblasDgetriBatched(cublasH, p, aconst, p, pivotArray, c, p, info, batch);
  if (info[0] < 0)
  {
    hipblasDestroy(cublasH);
    return info[0];
  }

  // X'Y   (p*n)*(n*1) = p*1
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           n, p,
                           &alpha,
                           d_X, n,
                           d_Y, 1,
                           &beta,
                           d_coef2, 1);

  // (X'X)^{-1}X'Y
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           p, p,
                           &alpha,
                           d_invXX, p,
                           d_coef2, 1,
                           &beta,
                           d_coef, 1);
  free(pivotArray);
  //free(info);
  free(d_XX);
  free(d_coef2);
  free(a);
  free(c);
  hipblasDestroy(cublasH);
  return 0;
}

__global__ void kernel(int n, int p,
  const double *d_X,
  const double *d_Y,
  double *d_invXX,
  double *d_coef)
{
  __syncthreads();
  solveBeta(int n, int p,
                            const double *d_X,
                            const double *d_Y,
                            double *d_invXX,
                            double *d_coef);
  __syncthreads();
}

int main(int argc, char const *argv[]) {
  /* code */
  //double A[] = {1, 2, 3, 0, 2, 4, 2, 1, 5};
  double A[] = {1, 1, 1, 1, 2, 3, 5, 4, 3, 6, 7, 9};
  double B[] = {1, 2, 3, 4};
  double *d_A, *d_B, *d_invXX, *d_coef;
  double coef[3];
  hipMalloc((void**)&d_A, sizeof(double)*12);
  hipMalloc((void**)&d_B, sizeof(double)*4);
  hipMalloc((void**)&d_invXX, sizeof(double)*9);
  hipMalloc((void**)&d_coef, sizeof(double)*3);
  hipMemcpy(d_A, A, sizeof(double)*12, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(double)*4, hipMemcpyHostToDevice);
  solveBeta<<<4, 4>>>(4, 3, d_A, d_B, d_invXX, d_coef);
  hipDeviceSynchronize();
  hipMemcpy(coef, d_coef, sizeof(double)*3, hipMemcpyDeviceToHost);
  for (int i = 0; i < 3; i++)
  {
    printf("%f, \n", coef[i]);
  }
  return 0;
}
