#include "hip/hip_runtime.h"

/* Includes, system */
#include <stdio.h>
#include <stdlib.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <gsl/gsl_rng.h>
#include <gsl/gsl_matrix.h>

__device__ int solveBeta(
                          int n, int p,
                          const double *d_X,
                          const double *d_Y,
                          double *d_invXX,
                          double *d_coef)
{
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = hipblasCreate(&cublasH);
  int *pivotArray = (int *)malloc(p*sizeof(int));
  int *info = (int *)malloc(sizeof(int));
  info[0] = 0;
  //int info;
  int batch = 1;
  double alpha = 1.0, beta = 0.0;

  double *d_XX = (double *)malloc(sizeof(double)*p*p);
  double *d_coef2 = (double *)malloc(sizeof(double)*p);

  double **a = (double **)malloc(sizeof(double *));
  *a = d_XX;
  const double **aconst = (const double **)a;

  double **c = (double **)malloc(sizeof(double *));
  *c = d_invXX;
  // X'X
  cublas_status = hipblasDgemm(cublasH,
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           p, p, n, // DO NOT mess up the order
                           &alpha,
                           d_X, n,
                           d_X, n,
                           &beta,
                           d_XX, p);

  // inv(X'X)
  cublas_status = hipblasDgetrfBatched(cublasH, p, a, p, pivotArray, info, batch);
  if (info[0] < 0)
  {
    hipblasDestroy(cublasH);
    return info[0];
  }
  else if (info[0] > 0)
  {
    hipblasDestroy(cublasH);
    return info[0];
  }
  cublas_status = hipblasDgetriBatched(cublasH, p, aconst, p, pivotArray, c, p, info, batch);
  if (info[0] < 0)
  {
    hipblasDestroy(cublasH);
    return info[0];
  }

  // X'Y   (p*n)*(n*1) = p*1
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           n, p,
                           &alpha,
                           d_X, n,
                           d_Y, 1,
                           &beta,
                           d_coef2, 1);

  // (X'X)^{-1}X'Y
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           p, p,
                           &alpha,
                           d_invXX, p,
                           d_coef2, 1,
                           &beta,
                           d_coef, 1);
  free(pivotArray);
  free(info);
  free(d_XX);
  free(d_coef2);
  free(a);
  free(c);
  hipblasDestroy(cublasH);
  return 0;
}

__global__ void kernel(int n, int p,
  const double *d_X,
  const double *d_Y,
  double *d_GinvXX,
  double *d_Gcoef)
{
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  //printf("%d\n", tid);
  double *d_coef = (double*)malloc(sizeof(double)*p);
  double *d_invXX = (double*)malloc(sizeof(double)*p*p);
  __syncthreads(); // must add
  solveBeta(n, p, d_X, d_Y, d_invXX, d_coef);
  __syncthreads(); // must add
  /*
  if (res != 0)
  {
    free(d_coef);
    free(d_invXX);
    return;
  }
  */
  __syncthreads(); // must add
  for (int i = 0; i < p; i++)
    d_Gcoef[tid*p+i] = d_coef[i];
  __syncthreads();
  free(d_coef);
  free(d_invXX);
}

int main(int argc, char const *argv[]) {
  /* code */
  //double A[] = {1, 2, 3, 0, 2, 4, 2, 1, 5};
  double A[] = {1, 1, 1, 1, 2, 3, 5, 4, 3, 6, 7, 9};
  double B[] = {1, 2, 3, 4};
  double *d_A, *d_B, *d_invXX, *d_coef;
  int threadsPerBlock = 256;
  int blocksPerGird = 1;
  double coef[3*threadsPerBlock*blocksPerGird];
  hipMalloc((void**)&d_A, sizeof(double)*12);
  hipMalloc((void**)&d_B, sizeof(double)*4);
  hipMalloc((void**)&d_invXX, sizeof(double)*9);
  hipMalloc((void**)&d_coef, sizeof(double)*3*threadsPerBlock*blocksPerGird);
  hipMemcpy(d_A, A, sizeof(double)*12, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(double)*4, hipMemcpyHostToDevice);
  kernel<<<threadsPerBlock, blocksPerGird>>>(4, 3, d_A, d_B, d_invXX, d_coef);
  hipDeviceSynchronize();
  hipMemcpy(coef, d_coef, sizeof(double)*3*threadsPerBlock*blocksPerGird, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(d_coef);
  hipFree(d_invXX);
  hipFree(d_A);
  hipFree(d_B);
  hipDeviceReset();

  for (int i = 0; i < threadsPerBlock*blocksPerGird; i++)
  {
    for (int j = 0; j < 3; j++)
      printf("%f, \n", coef[i*3+j]);
    printf("\n");
  }
  return 0;
}
