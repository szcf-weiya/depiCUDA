#include "hip/hip_runtime.h"

/* Includes, system */
#include <stdio.h>
#include <stdlib.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <gsl/gsl_rng.h>
#include <gsl/gsl_matrix.h>

__global__ void inverse(double* d_XX, int p, double* d_invXX)
{
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = hipblasCreate(&cublasH);
  int *pivotArray = (int *)malloc(p*sizeof(int));
  int *info = (int*)malloc(sizeof(int));
  info[0] = 0;
  //int info = 0;
  int batch = 1;

  /*
  if (info < 0)
  {
    hipblasDestroy(cublasH);
    return;
  }
  else if (info > 0)
  {
    hipblasDestroy(cublasH);
    return;
  }
  */

  double **a = (double **)malloc(sizeof(double *));
  *a = d_XX;
  const double **aconst = (const double **)a;
  double **c = (double **)malloc(sizeof(double *));
  *c = d_invXX;
  cublas_status = hipblasDgetrfBatched(cublasH, p, a, p, pivotArray, info, batch);
  cublas_status = hipblasDgetriBatched(cublasH, p, aconst, p, pivotArray, c, p, info, batch);
      /*
  if (info < 0)
  {
    hipblasDestroy(cublasH);
    return;
  }
  */
  hipblasDestroy(cublasH);
}

int main(int argc, char const *argv[]) {
  /* code */
  //double A[] = {1, 2, 3, 0, 2, 4, 2, 1, 5};
  double A[] = {1, 0, 0, 0, 1, 0, 0, 0, 1};
  double B[9];
  double *d_A, *d_B;
  hipMalloc((void**)&d_A, sizeof(double)*9);
  hipMalloc((void**)&d_B, sizeof(double)*9);
  hipMemcpy(d_A, A, sizeof(double)*9, hipMemcpyHostToDevice);
  inverse<<<1, 1>>>(d_A, 3, d_B);
  hipDeviceSynchronize();
  hipMemcpy(B, d_B, sizeof(double)*9, hipMemcpyDeviceToHost);
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
      printf("%f, ", B[i*3+j]);
    printf("\n");
  }
  return 0;
}
