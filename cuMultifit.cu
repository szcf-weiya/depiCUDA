#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

#include <gsl/gsl_matrix.h>
#include <gsl/gsl_vector.h>
#include <gsl/gsl_cdf.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "cuLUsolve.h"
#include "cuMultifit.h"

int cuMultifit(const double *X, int n, int p, const double *Y, double *coef, double *pvalue)
{
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
  hipError_t cudaStat = hipSuccess;

  const int lda = n;
  double *C;
  C = (double*)malloc(sizeof(double)*p*p);

  double *d_X = NULL;
  double *d_C = NULL;
  double *d_Y = NULL;
  double *d_Yhat = NULL;
  double *d_coef = NULL;
  double *d_coef2 = NULL;


  // create cublas handle
  cublas_status = hipblasCreate(&cublasH);
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

  // copy to device
  cudaStat = hipMalloc ((void**)&d_X, sizeof(double) * lda * p);
  assert(hipSuccess == cudaStat);

  cudaStat = hipMalloc ((void**)&d_C, sizeof(double) * p * p);
  assert(hipSuccess == cudaStat);

  cudaStat = hipMalloc ((void**)&d_Y, sizeof(double) * n);
  assert(hipSuccess == cudaStat);

  cudaStat = hipMalloc ((void**)&d_Yhat, sizeof(double) * n);
  assert(hipSuccess == cudaStat);

  cudaStat = hipMalloc ((void**)&d_coef, sizeof(double) * p);
  assert(hipSuccess == cudaStat);

  cudaStat = hipMalloc ((void**)&d_coef2, sizeof(double) * p);
  assert(hipSuccess == cudaStat);

  cudaStat = hipMemcpy(d_X, X, sizeof(double) * lda * p, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat);
  cudaStat = hipMemcpy(d_Y, Y, sizeof(double) * n, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat);
  cudaStat = hipMemcpy(d_Yhat, Y, sizeof(double) * n, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat);
  double alpha_v = 1.0;
  double beta_v = 0.0;
  double *alpha = &alpha_v, *beta = &beta_v; //check!!
  printf("%f\n", *alpha);
  // d_C = d_X^T d_X
  cublas_status = hipblasDgemm(cublasH,
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           p, p, n, // DO NOT mess up the order
                           alpha,
                           d_X, n,
                           d_X, n,
                           beta,
                           d_C, p);
  cudaStat = hipDeviceSynchronize();
  assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
  assert(hipSuccess == cudaStat);
  printf("finish X'X\n");
  // copy d_C to C
  cudaStat = hipMemcpy(C, d_C, sizeof(double)*p*p, hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat);
  // inv(C)
  gsl_matrix *B = gsl_matrix_alloc(p, p);
  gsl_matrix_set_identity(B);

  cuda_LU_solve(C, p, B->data, p);
  cudaStat = hipMemcpy(d_C, B->data, sizeof(double)*p*p, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat);
  for (int i = 0; i < p*p; i++)
    printf("%f\n", B->data[i]);

  printf("finish inv(C)\n");
  printf("%f %f\n", *alpha, *beta);
  // d_Y = d_X^T * d_Y
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           n, p,
                           alpha,
                           d_X, n,
                           d_Y, 1,
                           beta,
                           d_coef, 1);
  cudaStat = hipDeviceSynchronize();
  assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
  assert(hipSuccess == cudaStat);
  cudaStat = hipMemcpy(coef, d_coef, sizeof(double) * p, hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat);
  for (int i = 0 ; i < p ; i ++ )
    printf("%f\n", coef[i]);

  // inv(C) * d_Y
  // due to by-column in gpu while by-row in gsl, C need to be transpose
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_T,
                           p, p,
                           alpha,
                           d_C, p,
                           d_coef, 1,
                           beta,
                           d_coef2, 1);
  cudaStat = hipDeviceSynchronize();
  assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
  assert(hipSuccess == cudaStat);

  // rss
  beta_v = -1.0;
  cublas_status = hipblasDgemv(cublasH, HIPBLAS_OP_N,
                           n, p,
                           alpha,
                           d_X, n,
                           d_coef2, 1,
                           beta,
                           d_Yhat, 1);
  cudaStat = hipDeviceSynchronize();
  assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
  assert(hipSuccess == cudaStat);

  // sigma ^2 = RSS/(n-p-1)
  double sigma;
  double *psigma = &sigma;
  hipblasDnrm2(cublasH, n, d_Yhat, 1, psigma);
  sigma = sigma/sqrt(n-p);

  // copy to coef
  cudaStat = hipMemcpy(coef, d_coef2, sizeof(double) * p, hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat);

  double tscore;
  for (int i = 0; i < p; i++)
  {
    tscore = coef[i]/(sigma*sqrt(gsl_matrix_get(B, i, i)));
    pvalue[i] = 2*(tscore < 0 ? gsl_cdf_tdist_P(tscore, n-p) : gsl_cdf_tdist_P(-tscore, n-p));
  }

  gsl_matrix_free(B);
  hipFree(d_X);
  hipFree(d_Y);
  hipFree(d_Yhat);
  hipFree(d_C);
  hipFree(d_coef);
  hipFree(d_coef2);


  hipblasDestroy(cublasH);
  hipDeviceReset();
  return 0;
}
